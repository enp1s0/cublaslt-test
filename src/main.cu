
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblaslt.h>

template <class T>
void eval(const std::size_t N) {
  T *dev_a, *dev_b, *dev_c;
  hipMallocManaged(&dev_a, sizeof(T) * N * N);
  hipMallocManaged(&dev_b, sizeof(T) * N * N);
  hipMallocManaged(&dev_c, sizeof(T) * N * N);

#pragma omp parallel for
  for (std::size_t i = 0; i < N * N; i++) {
    dev_a[i] = i;
    dev_b[i] = i + 3;
    dev_c[i] = i + 2;
  }

  hipblasLtHandle_t handle;
  hipblasLtCreate(&handle);

  hipblasLtMatmulDesc_t op_desc = nullptr;
  hipblasLtMatrixLayout_t desc_a = nullptr, desc_b = nullptr, desc_c = nullptr;
  hipblasLtMatmulPreference_t preference = nullptr;

  hipblasOperation_t trans_a, trans_b;

  hipblasLtMatmulDescCreate(&op_desc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  hipblasLtMatmulDescSetAttribute(op_desc, HIPBLASLT_MATMUL_DESC_TRANSA, &trans_a, sizeof(trans_a));
  hipblasLtMatmulDescSetAttribute(op_desc, HIPBLASLT_MATMUL_DESC_TRANSB, &trans_b, sizeof(trans_b));

  hipblasLtMatrixLayoutCreate(&desc_a, HIP_R_32F, N, N, N);
  hipblasLtMatrixLayoutCreate(&desc_b, HIP_R_32F, N, N, N);
  hipblasLtMatrixLayoutCreate(&desc_c, HIP_R_32F, N, N, N);

  hipblasLtMatmulPreferenceCreate(&preference);
  std::size_t workspace_size;
  hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspace_size, sizeof(workspace_size));
  void* workspace;
  hipMalloc(&workspace, 4lu << 20);

  int returned_results = 0;
  hipblasLtMatmulHeuristicResult_t heuristic_result = {};
  hipblasLtMatmulAlgoGetHeuristic(handle, op_desc, desc_a, desc_b, desc_c, desc_c, preference, 1, &heuristic_result, &returned_results);

  const T alpha = 1, beta = 0;
  hipblasLtMatmul(
      handle,
      op_desc,
      &alpha,
      dev_a, desc_a,
      dev_b, desc_b,
      &beta,
      desc_c, desc_c,
      desc_c, desc_c,
      &heuristic_result.algo,
      workspace,
      workspace_size,
      0
      );

  hipblasLtMatmulPreferenceDestroy(preference);
  hipblasLtMatrixLayoutDestroy(desc_a);
  hipblasLtMatrixLayoutDestroy(desc_b);
  hipblasLtMatrixLayoutDestroy(desc_c);
  hipblasLtMatmulDescDestroy(op_desc);

  hipblasLtDestroy(handle);

  hipFree(workspace);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
}

int main() {
  eval<float>(1lu << 14);
}
