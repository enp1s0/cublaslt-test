#include <iostream>
#include <sstream>
#include <hipblaslt.h>
#include <stdexcept>

inline void cuda_check_error(const hipError_t error, const std::string filename, const std::size_t line, const std::string funcname, const std::string message = ""){
  if(error != hipSuccess){
    std::stringstream ss;
    ss << hipGetErrorString( error );
    if(message.length() != 0){
      ss << " : " << message;
    }
    ss << " [" << filename << ":" << line << " in " << funcname << "]";
    throw std::runtime_error(ss.str());
  }
}

inline void cuda_check_error(const hipblasStatus_t error, const std::string filename, const std::size_t line, const std::string funcname, const std::string message = ""){
  if(error != HIPBLAS_STATUS_SUCCESS){
    std::stringstream ss;
    ss << cublasGetStatusString(error);
    if(message.length() != 0){
      ss << " : " << message;
    }
    ss << " [" << filename << ":" << line << " in " << funcname << "]";
    throw std::runtime_error(ss.str());
  }
}
#ifndef CUDA_CHECK_ERROR
#define CUDA_CHECK_ERROR(status) cuda_check_error(status, __FILE__, __LINE__, __func__)
#endif


template <class T>
void eval(const std::size_t N) {
  T *dev_a, *dev_b, *dev_c;
  CUDA_CHECK_ERROR(hipMallocManaged(&dev_a, sizeof(T) * N * N));
  CUDA_CHECK_ERROR(hipMallocManaged(&dev_b, sizeof(T) * N * N));
  CUDA_CHECK_ERROR(hipMallocManaged(&dev_c, sizeof(T) * N * N));

#pragma omp parallel for
  for (std::size_t i = 0; i < N * N; i++) {
    dev_a[i] = i;
    dev_b[i] = i + 3;
    dev_c[i] = i + 2;
  }

  CUDA_CHECK_ERROR(hipMemAdvise(dev_a, sizeof(T) * N * N, hipMemAdviseSetAccessedBy, 0));
  CUDA_CHECK_ERROR(hipMemAdvise(dev_b, sizeof(T) * N * N, hipMemAdviseSetAccessedBy, 0));
  CUDA_CHECK_ERROR(hipMemAdvise(dev_c, sizeof(T) * N * N, hipMemAdviseSetAccessedBy, 0));

  hipblasLtHandle_t handle;
  CUDA_CHECK_ERROR(hipblasLtCreate(&handle));

  hipblasLtMatmulDesc_t op_desc = nullptr;
  hipblasLtMatrixLayout_t desc_a = nullptr, desc_b = nullptr, desc_c = nullptr;
  hipblasLtMatmulPreference_t preference = nullptr;

  const hipblasOperation_t trans_a = HIPBLAS_OP_N, trans_b = HIPBLAS_OP_N;

  CUDA_CHECK_ERROR(hipblasLtMatmulDescCreate(&op_desc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
  CUDA_CHECK_ERROR(hipblasLtMatmulDescSetAttribute(op_desc, HIPBLASLT_MATMUL_DESC_TRANSA, &trans_a, sizeof(trans_a)));
  CUDA_CHECK_ERROR(hipblasLtMatmulDescSetAttribute(op_desc, HIPBLASLT_MATMUL_DESC_TRANSB, &trans_b, sizeof(trans_b)));

  CUDA_CHECK_ERROR(hipblasLtMatrixLayoutCreate(&desc_a, HIP_R_32F, N, N, N));
  CUDA_CHECK_ERROR(hipblasLtMatrixLayoutCreate(&desc_b, HIP_R_32F, N, N, N));
  CUDA_CHECK_ERROR(hipblasLtMatrixLayoutCreate(&desc_c, HIP_R_32F, N, N, N));

  CUDA_CHECK_ERROR(hipblasLtMatmulPreferenceCreate(&preference));
  std::size_t workspace_size = 4lu << 20;
  CUDA_CHECK_ERROR(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspace_size, sizeof(workspace_size)));
  void* workspace;
  CUDA_CHECK_ERROR(hipMalloc(&workspace, workspace_size));

  int returned_results = 0;
  hipblasLtMatmulHeuristicResult_t heuristic_result = {};
  CUDA_CHECK_ERROR(hipblasLtMatmulAlgoGetHeuristic(handle, op_desc, desc_a, desc_b, desc_c, desc_c, preference, 1, &heuristic_result, &returned_results));

  const T alpha = 1, beta = 0;
  CUDA_CHECK_ERROR(hipblasLtMatmul(
      handle,
      op_desc,
      &alpha,
      dev_a, desc_a,
      dev_b, desc_b,
      &beta,
      dev_c, desc_c,
      dev_c, desc_c,
      &heuristic_result.algo,
      workspace,
      workspace_size,
      0
      ));
  CUDA_CHECK_ERROR(hipDeviceSynchronize());

  CUDA_CHECK_ERROR(hipblasLtMatmulPreferenceDestroy(preference));
  CUDA_CHECK_ERROR(hipblasLtMatrixLayoutDestroy(desc_a));
  CUDA_CHECK_ERROR(hipblasLtMatrixLayoutDestroy(desc_b));
  CUDA_CHECK_ERROR(hipblasLtMatrixLayoutDestroy(desc_c));
  CUDA_CHECK_ERROR(hipblasLtMatmulDescDestroy(op_desc));

  CUDA_CHECK_ERROR(hipblasLtDestroy(handle));

  CUDA_CHECK_ERROR(hipFree(workspace));
  CUDA_CHECK_ERROR(hipFree(dev_a));
  CUDA_CHECK_ERROR(hipFree(dev_b));
  CUDA_CHECK_ERROR(hipFree(dev_c));
}

int main() {
  eval<float>(1lu << 14);
}
